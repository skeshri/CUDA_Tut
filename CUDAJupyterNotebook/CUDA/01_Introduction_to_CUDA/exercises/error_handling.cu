
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel (int* a) {
    a[-1] = 1;
}

int main() {
    int* a;
    hipMalloc(&a, -sizeof(int));

    kernel<<<1, -1>>>(a);

    hipDeviceSynchronize();

    free(a);
}
