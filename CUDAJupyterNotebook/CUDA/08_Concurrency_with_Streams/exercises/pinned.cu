
#include <hip/hip_runtime.h>
#include <cstdio>

#define N (1024 * 1024 * 32)

__global__ void setval (int* a) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        a[idx] = 1;
    }
}

int main() {

    // The first CUDA call pays for CUDA context creation;
    // insert a non-functional call here to pay for that cost
    // so we can better understand the true cost of the memory
    // allocation below.
    hipFree(0);

    int* h_a;
    int* d_a;

    h_a = (int*) malloc(N * sizeof(int));
    hipMalloc(&d_a, N * sizeof(int));

    memset(h_a, 0, N * sizeof(int));

    hipMemcpy(d_a, h_a, N * sizeof(int), hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (N + threads - 1) / threads;

    setval<<<blocks, threads>>>(d_a);
    hipDeviceSynchronize();

    hipMemcpy(h_a, d_a, N * sizeof(int), hipMemcpyDeviceToHost);

    // Verify the data got updated as we expect.
    if (h_a[0] == 1) {
        printf("Success!\n");
    }
    else {
        printf("Failure!\n");
    }

    free(h_a);
    hipFree(d_a);

    return 0;

}
