
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (32*1048576)

__global__ void reduce (double* A, double* sum) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        atomicAdd(sum, A[idx]);
    }
}

int main()
{
    double *h_A;
    double* h_sum;
    h_A = new double[N];
    h_sum = new double;

    double* d_A;
    double* d_sum;
    hipMalloc(&d_A, N * sizeof(double));
    hipMalloc(&d_sum, sizeof(double));

    // initialize data in host memory
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
    }
    *h_sum = 0.0f;

    // copy data to device memory
    hipMemcpy(d_A, h_A, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_sum, h_sum, sizeof(double), hipMemcpyHostToDevice);

    // sum the array on the device
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    
    reduce<<<blocks, threads>>>(d_A, d_sum);
    hipDeviceSynchronize();

    // check the device results
    hipMemcpy(h_sum, d_sum, sizeof(double), hipMemcpyDeviceToHost);
    if (*h_sum != (double) N) {
        printf("device reduction incorrect!\n");
        return -1;
    }
    printf("device reduction correct!\n");

    free(h_A);
    free(h_sum);
    hipFree(d_A);
    hipFree(d_sum);

    return 0;
}
